#include "hip/hip_runtime.h"
#include<iostream>
#include<opencv2/core.hpp>
#include<opencv2/imgcodecs.hpp>
#include<opencv2/highgui.hpp>
#include <boost/program_options.hpp>

using namespace std;
using namespace cv;
using namespace boost::program_options;

__global__ void rgbtogray(unsigned char *input, unsigned char *output, int numRows, int numCols, int step) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= numCols || row >= numRows) return;

    int tid = row * step + 3 * col;
    unsigned char r = input[tid];
    unsigned char g = input[tid + 1];
    unsigned char b = input[tid + 2];

    output[row * numCols + col] = static_cast<unsigned char>(r * 0.299f + g * 0.587f + b * 0.114f);
}

__global__ void mirrorImage(unsigned char* input, unsigned char* output, int numRows, int numCols, int channels, int step) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= numCols || row >= numRows) return;

    int tid = row * step + (channels * col);
    int tid_new = row * step + (channels * (numCols - col - 1));

    for(int i = 0; i < channels; i++)
        output[tid_new + i] = input[tid + i];
}

void checkCudaError(hipError_t result, const char *function) {
    if (result != hipSuccess) {
        cerr << "CUDA Error in " << function << ": " << hipGetErrorString(result) << endl;
        exit(-1);
    }
}

int main(int argc, char **argv) {
    options_description desc("Allowed Options");
    desc.add_options()
        ("help,h", "Display help screen")
        ("output,o", value<string>()->default_value("output.jpg"), "Specify output file name")
        ("mirror,m", "Mirror the image")
        ("gray,g", "RGB to grayscale conversion");

    variables_map vm;
    store(parse_command_line(argc, argv, desc), vm);
    notify(vm);

    if(vm.count("help") || argc < 2){
        cout << desc << "\n";
        return 1;
    }

    string input_file = argv[1];
    string output_file = vm["output"].as<string>();

    Mat input = imread(input_file, IMREAD_COLOR);
    if(input.empty()) {
        cerr << "Image Not Found: " << input_file << endl;
        return -1;
    }

    Mat output(input.rows, input.cols, input.type());
    dim3 block_size(16, 16);
    dim3 num_blocks((input.cols + block_size.x - 1) / block_size.x, (input.rows + block_size.y - 1) / block_size.y);

    unsigned char *d_input, *d_output;
    size_t numBytes = input.step * input.rows;
    checkCudaError(hipMalloc<unsigned char>(&d_input, numBytes), "hipMalloc d_input");
    checkCudaError(hipMalloc<unsigned char>(&d_output, numBytes), "hipMalloc d_output");
    checkCudaError(hipMemcpy(d_input, input.ptr(), numBytes, hipMemcpyHostToDevice), "hipMemcpy HostToDevice");

    if(vm.count("mirror")) {
        mirrorImage<<<num_blocks, block_size>>>(d_input, d_output, input.rows, input.cols, input.channels(), input.step);
    } else if(vm.count("gray")) {
        rgbtogray<<<num_blocks, block_size>>>(d_input, d_output, input.rows, input.cols, input.step);
    }

    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");
    checkCudaError(hipMemcpy(output.ptr(), d_output, numBytes, hipMemcpyDeviceToHost), "hipMemcpy DeviceToHost");

    hipFree(d_input);
    hipFree(d_output);

    imwrite(output_file, output);

    return 0;
}